#include "hip/hip_runtime.h"
// nbody_cuda.cu - CUDA implementation file
#include "nbody_cuda.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <math.h>

// Constant memory for physical constants
__constant__ double d_G;

// CUDA kernel to calculate forces between bodies
__global__ void calculateForces(Body* bodies, int numBodies) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < numBodies) {
        // Reset acceleration
        bodies[idx].ax = bodies[idx].ay = bodies[idx].az = 0.0;
        
        // Calculate forces from all other bodies
        for (int j = 0; j < numBodies; j++) {
            if (idx != j) {
                double dx = bodies[j].x - bodies[idx].x;
                double dy = bodies[j].y - bodies[idx].y;
                double dz = bodies[j].z - bodies[idx].z;
                
                double distSq = dx*dx + dy*dy + dz*dz;
                double dist = sqrt(distSq + 1e-10); // Softening to avoid division by zero
                
                double F = d_G * bodies[idx].m * bodies[j].m / (distSq + 1e-10);
                
                // Force components
                double Fx = F * dx / dist;
                double Fy = F * dy / dist;
                double Fz = F * dz / dist;
                
                // Accumulate acceleration
                bodies[idx].ax += Fx / bodies[idx].m;
                bodies[idx].ay += Fy / bodies[idx].m;
                bodies[idx].az += Fz / bodies[idx].m;
            }
        }
    }
}

// CUDA kernel to update positions and velocities
__global__ void integratePositions(Body* bodies, int numBodies, double dt) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < numBodies) {
        // Update velocity
        bodies[idx].vx += bodies[idx].ax * dt;
        bodies[idx].vy += bodies[idx].ay * dt;
        bodies[idx].vz += bodies[idx].az * dt;
        
        // Update position (with semi-implicit Euler integration)
        bodies[idx].x += bodies[idx].vx * dt + 0.5 * bodies[idx].ax * dt * dt;
        bodies[idx].y += bodies[idx].vy * dt + 0.5 * bodies[idx].ay * dt * dt;
        bodies[idx].z += bodies[idx].vz * dt + 0.5 * bodies[idx].az * dt * dt;
    }
}

// Helper function to check for collisions on GPU and mark bodies for merging
__global__ void detectCollisions(Body* bodies, int* collisionMap, int numBodies) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < numBodies && collisionMap[idx] == -1) {
        for (int j = idx + 1; j < numBodies; j++) {
            if (collisionMap[j] == -1) { // Only check non-merged bodies
                double dx = bodies[idx].x - bodies[j].x;
                double dy = bodies[idx].y - bodies[j].y;
                double dz = bodies[idx].z - bodies[j].z;
                
                double distSq = dx*dx + dy*dy + dz*dz;
                double radiusSum = bodies[idx].radius + bodies[j].radius;
                
                if (distSq < radiusSum * radiusSum) {
                    // Mark the smaller body for merging into the larger one
                    if (bodies[idx].m < bodies[j].m) {
                        collisionMap[idx] = j;
                    } else {
                        collisionMap[j] = idx;
                    }
                }
            }
        }
    }
}

// Implementation of the CUDA step function
void cudaStep(Body* h_bodies, int& numBodies, double dt) {
    // Allocate device memory
    Body* d_bodies;
    int* d_collisionMap;
    hipMalloc((void**)&d_bodies, numBodies * sizeof(Body));
    hipMalloc((void**)&d_collisionMap, numBodies * sizeof(int));
    
    // Copy data to device
    hipMemcpy(d_bodies, h_bodies, numBodies * sizeof(Body), hipMemcpyHostToDevice);
    
    // Set physical constants in constant memory
    double h_G = 6.67430e-11;
    hipMemcpyToSymbol(HIP_SYMBOL(d_G), &h_G, sizeof(double));
    
    // Initialize collision map to -1 (no collisions)
    int* h_collisionMap = new int[numBodies];
    for (int i = 0; i < numBodies; i++) {
        h_collisionMap[i] = -1;
    }
    hipMemcpy(d_collisionMap, h_collisionMap, numBodies * sizeof(int), hipMemcpyHostToDevice);
    
    // Calculate number of CUDA blocks and threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (numBodies + threadsPerBlock - 1) / threadsPerBlock;
    
    // Execute kernels
    calculateForces<<<blocksPerGrid, threadsPerBlock>>>(d_bodies, numBodies);
    integratePositions<<<blocksPerGrid, threadsPerBlock>>>(d_bodies, numBodies, dt);
    detectCollisions<<<blocksPerGrid, threadsPerBlock>>>(d_bodies, d_collisionMap, numBodies);
    
    // Synchronize to ensure all kernels have completed
    hipDeviceSynchronize();
    
    // Copy results back to host
    hipMemcpy(h_bodies, d_bodies, numBodies * sizeof(Body), hipMemcpyDeviceToHost);
    hipMemcpy(h_collisionMap, d_collisionMap, numBodies * sizeof(int), hipMemcpyDeviceToHost);
    
    // Handle collisions and mergers on CPU
    int newNumBodies = numBodies;
    for (int i = 0; i < numBodies; i++) {
        if (h_collisionMap[i] != -1) {
            int targetIdx = h_collisionMap[i];
            
            // Conservation of momentum
            h_bodies[targetIdx].vx = (h_bodies[targetIdx].m * h_bodies[targetIdx].vx + h_bodies[i].m * h_bodies[i].vx) / 
                                     (h_bodies[targetIdx].m + h_bodies[i].m);
            h_bodies[targetIdx].vy = (h_bodies[targetIdx].m * h_bodies[targetIdx].vy + h_bodies[i].m * h_bodies[i].vy) / 
                                     (h_bodies[targetIdx].m + h_bodies[i].m);
            h_bodies[targetIdx].vz = (h_bodies[targetIdx].m * h_bodies[targetIdx].vz + h_bodies[i].m * h_bodies[i].vz) / 
                                     (h_bodies[targetIdx].m + h_bodies[i].m);
            
            // Combine masses
            h_bodies[targetIdx].m += h_bodies[i].m;
            
            // Update radius (assuming volume conservation)
            h_bodies[targetIdx].radius = pow(pow(h_bodies[targetIdx].radius, 3) + pow(h_bodies[i].radius, 3), 1.0/3.0);
            
            // Mark this body for removal
            h_bodies[i].m = 0;
            newNumBodies--;
        }
    }
    
    // Compact the array to remove merged bodies
    if (newNumBodies < numBodies) {
        int writeIdx = 0;
        for (int i = 0; i < numBodies; i++) {
            if (h_bodies[i].m > 0) {
                if (writeIdx != i) {
                    h_bodies[writeIdx] = h_bodies[i];
                }
                writeIdx++;
            }
        }
        numBodies = newNumBodies;
    }
    
    // Clean up
    hipFree(d_bodies);
    hipFree(d_collisionMap);
    delete[] h_collisionMap;
}
